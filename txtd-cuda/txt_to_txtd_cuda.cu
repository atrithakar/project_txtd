#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <hip/hip_runtime.h>
#include <openssl/sha.h>

#if defined(_WIN32)
#include <direct.h>
#define MKDIR(dir) _mkdir(dir)
#else
#include <sys/stat.h>
#define MKDIR(dir) mkdir(dir, 0777)
#endif

#define CHUNK_SIZE 500  // Number of bytes per GPU thread (change as needed)

// Host-side encode_char table
__device__ __constant__ unsigned char encode_table[128] = {
    // ...fill with 0xFF for all, then set valid chars below...
    // 0-9, ., space, tab, newline, comma, '\0'
    // ASCII: '0' = 48, '1' = 49, ..., '9' = 57, '.' = 46, ' ' = 32, '\t' = 9, '\n' = 10, ',' = 44
    // e.g. encode_table['0'] = 0b0000; encode_table['1'] = 0b0001; etc.
};

__global__ void encode_kernel(const char *in, unsigned char *out, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t start = idx * CHUNK_SIZE;
    size_t end = start + CHUNK_SIZE;
    if (start >= n) return;
    if (end > n) end = n;
    for (size_t i = start; i < end; ++i) {
        char c = in[i];
        unsigned char code = (c >= 0 && c < 128) ? encode_table[(int)c] : 0xFF;
        out[i] = code;
    }
}

void fill_encode_table(unsigned char *table) {
    for (int i = 0; i < 128; ++i) table[i] = 0xFF;
    table['0'] = 0b0000; table['1'] = 0b0001; table['2'] = 0b0010; table['3'] = 0b0011;
    table['4'] = 0b0100; table['5'] = 0b0101; table['6'] = 0b0110; table['7'] = 0b0111;
    table['8'] = 0b1000; table['9'] = 0b1001; table['.'] = 0b1010; table[' '] = 0b1011;
    table['\t'] = 0b1100; table['\n'] = 0b1101; table[','] = 0b1110; table['\0'] = 0b1111;
}

// Helper to compute SHA-256 checksum and write as hex string to file
int write_checksum(const char *input_filename, const char *output_filename) {
    FILE *in = fopen(input_filename, "rb");
    if (!in) return 1;
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    unsigned char buf[32768];
    size_t n;
    while ((n = fread(buf, 1, sizeof(buf), in)) > 0)
        SHA256_Update(&sha256, buf, n);
    fclose(in);
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_Final(hash, &sha256);
    FILE *out = fopen(output_filename, "w");
    if (!out) return 2;
    for (int i = 0; i < SHA256_DIGEST_LENGTH; i++)
        fprintf(out, "%02x", hash[i]);
    fprintf(out, "\n");
    fclose(out);
    return 0;
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <input.txt>\n", argv[0]);
        return 1;
    }
    char *input_path = argv[1];
    char basename[256];
    const char *slash = strrchr(input_path, '/');
    #ifdef _WIN32
    const char *bslash = strrchr(input_path, '\\');
    if (!slash || (bslash && bslash > slash)) slash = bslash;
    #endif
    const char *start = slash ? slash + 1 : input_path;
    const char *dot = strrchr(start, '.');
    size_t len = dot ? (size_t)(dot - start) : strlen(start);
    strncpy(basename, start, len); basename[len] = '\0';

    // Create output directory named after input file (without extension)
    MKDIR(basename);

    char outpath[512], checksum_path[512];
    snprintf(outpath, sizeof(outpath), "%s/%s.encoded.txtd", basename, basename);
    snprintf(checksum_path, sizeof(checksum_path), "%s/%s.checksum.txt", basename, basename);

    // Read input file
    FILE *in = fopen(input_path, "rb");
    if (!in) { perror("open input"); return 1; }
    fseek(in, 0, SEEK_END);
    size_t n = ftell(in);
    fseek(in, 0, SEEK_SET);
    char *buf = (char*)malloc(n);
    fread(buf, 1, n, in);
    fclose(in);

    // CUDA encode
    char *d_in; unsigned char *d_out;
    hipMalloc(&d_in, n); hipMalloc(&d_out, n);
    hipMemcpy(d_in, buf, n, hipMemcpyHostToDevice);

    unsigned char table[128]; fill_encode_table(table);
    hipMemcpyToSymbol(HIP_SYMBOL(encode_table), table, 128);

    int threads_per_block = 256;
    int num_threads = (n + CHUNK_SIZE - 1) / CHUNK_SIZE;
    int num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;
    encode_kernel<<<num_blocks, threads_per_block>>>(d_in, d_out, n);

    unsigned char *codes = (unsigned char*)malloc(n);
    hipMemcpy(codes, d_out, n, hipMemcpyDeviceToHost);

    // Pack nibbles into bytes
    FILE *out = fopen(outpath, "wb");
    unsigned char byte = 0;
    int half = 0;
    for (size_t i = 0; i < n; ++i) {
        if (codes[i] == 0xFF) { fprintf(stderr, "Invalid char at %zu\n", i); exit(1); }
        if (half == 0) { byte = codes[i] << 4; half = 1; }
        else { byte |= codes[i]; fwrite(&byte, 1, 1, out); half = 0; }
    }
    if (half == 0) { byte = 0b1111 << 4; fwrite(&byte, 1, 1, out); }
    else { byte |= 0b1111; fwrite(&byte, 1, 1, out); }
    fclose(out);

    // Write checksum of original input file
    if (write_checksum(input_path, checksum_path) != 0) {
        fprintf(stderr, "Failed to write checksum file\n");
        // continue anyway
    }

    hipFree(d_in); hipFree(d_out); free(buf); free(codes);

    printf("Encoded file: %s\nChecksum file: %s\n", outpath, checksum_path);
    return 0;
}
